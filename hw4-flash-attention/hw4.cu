#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <sys/time.h>
#include <unistd.h>

#include <cassert>
#include <cfloat>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <numeric>

#ifdef PROFILING
#include <nvtx3/nvtx3.hpp>
#define NVTX_RANGE_START(arg) \
    nvtxRangePushA(#arg);
#define NVTX_RANGE_END() \
    nvtxRangePop();
#define NVTX_RANGE_FUNC() \
    NVTX3_FUNC_RANGE()
#else
#define NVTX_RANGE_START(arg) \
    {}
#define NVTX_RANGE_END() \
    {}
#define NVTX_RANGE_FUNC() \
    {}
#endif  // PROFILING

#define CUDA_CHECK(condition)                                                                                     \
    if ((condition) != hipSuccess) {                                                                             \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__, __LINE__); \
        exit(1);                                                                                                  \
    }

constexpr float FLOAT_MIN = -FLT_MAX;

struct Data {
    char *input_filename;
    char *output_filename;
    FILE *input_file;
    FILE *output_file;
    int B, N, d;
    float *O;
};

template <typename T>
void cuda_init_array(T *arr, size_t size, T val, hipStream_t stream);
template <typename T>
__global__ void cuda_init_array_kernel(T *arr, size_t size, T val);

namespace flash_attention {
void flash_attention_switch(Data *data);
template <int bc, int br, int cr, int bd, int num_warps, int threads_per_warp>
void flash_attention(Data *data);
template <int bc, int br, int cr, int bd, int num_warps, int threads_per_warp>
__global__ void flash_attention_kernel(float *O, float *Q, float *K, float *V, float *L, int N, int d);
template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void qk_dot_and_scalar(float *out, float *q, float *k, int d, float scalar);
template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_max(float *mij1, float *sij, float *mij0, int n);
template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void minus_max_and_exp(float *pij, float *sij, float *mij1);
template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_sum(float *lij1, float *pij, float *lij0, float *mij0, float *mij1, int n);
template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void inner_update_o(float *oi, float *pij, float *vj, float *mij0, float *mij1, int n, int d);
template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void outer_update_lo(float *lij1, float *oi, float *mij0, float *lij0, int d);
};  // namespace flash_attention

int main(int argc, char *argv[]) {
    NVTX_RANGE_FUNC();
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }

    Data data;

    data.input_filename = argv[1];
    data.output_filename = argv[2];

#ifdef PROFILING
    timespec ts;
    double start, end;
    clock_gettime(CLOCK_REALTIME, &ts);
    start = ts.tv_sec + ts.tv_nsec * 1e-9;
#endif  // PROFILING
    flash_attention::flash_attention_switch(&data);
#ifdef PROFILING
    clock_gettime(CLOCK_REALTIME, &ts);
    end = ts.tv_sec + ts.tv_nsec * 1e-9;
    fprintf(stderr, "took: %lf\n", end - start);
#endif  // PROFILING

    return 0;
}

template <typename T>
void cuda_init_array(T *arr, size_t size, T val, hipStream_t stream) {
    cuda_init_array_kernel<<<(int)ceil((float)size / 1024), 1024, 0, stream>>>(arr, size, val);
}

template <typename T>
__global__ void cuda_init_array_kernel(T *arr, size_t size, T val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        arr[idx] = val;
    }
}

namespace flash_attention {
void flash_attention_switch(Data *data) {
    data->input_file = fopen(data->input_filename, "rb");
    data->output_file = fopen(data->output_filename, "wb");
    fread(&data->B, sizeof(int), 1, data->input_file);
    fread(&data->N, sizeof(int), 1, data->input_file);
    fread(&data->d, sizeof(int), 1, data->input_file);
    if (data->d <= 32) {
        flash_attention<32, 32, 1, 37, 8, 16>(data);
    } else if (data->d <= 64) {
        flash_attention<32, 32, 1, 71, 8, 32>(data);
    }

    fclose(data->input_file);
    fclose(data->output_file);

#ifndef NO_FINALIZE
    hipHostFree(data->O);
#endif  // NO_FINALIZE
}

template <int bc, int br, int cr, int bd, int num_warps, int threads_per_warp>
void flash_attention(Data *data) {
    NVTX_RANGE_FUNC();
    int B = data->B;
    int N = data->N;
    int d = data->d;
#ifdef PROFILING
    fprintf(stderr, "B: %d, N: %d, d: %d\n", B, N, d);
#endif  // PROFILING
    int bb = (int)ceilf((float)B / 64);

    // Create a CUDA stream for asynchronous operations
    int num_streams = (int)ceil((float)B / bb);
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    float *Q, *K, *V, *O;
    hipHostMalloc(&Q, B * N * d * sizeof(float));
    hipHostMalloc(&K, B * N * d * sizeof(float));
    hipHostMalloc(&V, B * N * d * sizeof(float));
    hipHostMalloc(&O, B * N * d * sizeof(float));
    data->O = O;

    float *d_Q, *d_K, *d_V, *d_O;
    float *d_L;
    hipMalloc(&d_Q, B * N * d * sizeof(float));
    hipMalloc(&d_K, B * N * d * sizeof(float));
    hipMalloc(&d_V, B * N * d * sizeof(float));
    hipMalloc(&d_O, B * N * d * sizeof(float));
    hipMalloc(&d_L, B * N * sizeof(float));

    // Kernel launch
    const int smem_size = (br * bd +
                           br * bd +
                           bc * bd +
                           bc * bd +
                           br +
                           br +
                           br +
                           br +
                           br * bc +
                           br * bc) *
                          sizeof(float);

    NVTX_RANGE_START(flash_attention_execute);
    NVTX_RANGE_START(flash_attention_declare);
    for (int i = 0; i < num_streams; i++) {
        int num_batches = min(bb, B - i * bb);

        // Load data to host memory
        for (int j = 0; j < num_batches; j++) {
            fread(Q + (i * bb + j) * N * d, sizeof(float), N * d, data->input_file);
            fread(K + (i * bb + j) * N * d, sizeof(float), N * d, data->input_file);
            fread(V + (i * bb + j) * N * d, sizeof(float), N * d, data->input_file);
        }

        // Asynchronous memory copy and initialization
        hipMemcpyAsync(d_Q + i * bb * N * d, Q + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_K + i * bb * N * d, K + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_V + i * bb * N * d, V + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        // Kernel launch
        dim3 grid((int)ceilf((float)N / (br * cr)), num_batches);
        dim3 block(num_warps * threads_per_warp);
        flash_attention_kernel<bc, br, cr, bd, num_warps, threads_per_warp><<<grid, block, smem_size, streams[i]>>>(
            d_O + i * bb * N * d,
            d_Q + i * bb * N * d,
            d_K + i * bb * N * d,
            d_V + i * bb * N * d,
            d_L + i * bb * N,
            N, d);

        // Asynchronous memory copy back to host
        hipMemcpyAsync(O + i * bb * N * d, d_O + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }
    NVTX_RANGE_END();  // flash_attention_declare

    // Synchronize the stream to make sure all operations complete
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
        fwrite(data->O + i * bb * N * d, sizeof(float), bb * N * d, data->output_file);
    }
    NVTX_RANGE_END();  // flash_attention_execute

    // Clean up
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }

#ifndef NO_FINALIZE
    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_O);
    hipFree(d_L);

    hipHostFree(Q);
    hipHostFree(K);
    hipHostFree(V);
#endif  // NO_FINALIZE
}

template <int bc, int br, int cr, int bd, int num_warps, int threads_per_warp>
__global__ void flash_attention_kernel(float *O, float *Q, float *K, float *V, float *L, int N, int d) {
    // Thread and block index
    const int tx = threadIdx.x % num_warps;
    const int ty = threadIdx.x / num_warps;
    const int tc = (int)ceilf((float)N / bc);

    // Shared memory allocation
    extern __shared__ float shared_mem[];
    float *oi = shared_mem;      // (br, bd)
    float *qi = oi + br * bd;    // (br, bd)
    float *kj = qi + br * bd;    // (bc, bd)
    float *vj = kj + bc * bd;    // (bc, bd)
    float *lij0 = vj + bc * bd;  // (br)
    float *lij1 = lij0 + br;     // (br)
    float *mij0 = lij1 + br;     // (br)
    float *mij1 = mij0 + br;     // (br)
    float *sij = mij1 + br;      // (br, bc)
    float *pij = sij + br * bc;  // (br, bc)

    float *tmpptr;

    // Pointer to global memory
    float *o = O + blockIdx.y * N * d + blockIdx.x * cr * br * d;  // (cr, br, d)
    float *q = Q + blockIdx.y * N * d + blockIdx.x * cr * br * d;  // (cr, br, d)
    float *k = K + blockIdx.y * N * d;                             // (N, d)
    float *v = V + blockIdx.y * N * d;                             // (N, d)
    float *l = L + blockIdx.y * N + blockIdx.x * cr * br;          // (cr, br)

    float scalar = 1.0 / sqrtf(d);

    // Load O, Q, l, m to shared memory
    for (int y = ty; y < br; y += threads_per_warp) {
        for (int x = tx; x < d; x += num_warps) {
            oi[y * bd + x] = o[y * d + x];
            qi[y * bd + x] = q[y * d + x];
        }
    }
    if (threadIdx.x < br) {
        lij0[threadIdx.x] = 0;
#ifndef NO_ROWMAX
        mij0[threadIdx.x] = FLOAT_MIN;
#endif  // NO_ROWMAX
    }
    for (int j = 0; j < tc; j++) {
        int n = min(N - j * bc, bc);
        // Load K and V to shared memory
        for (int x = tx; x < bc; x += num_warps) {
            for (int y = ty; y < d; y += threads_per_warp) {
                kj[x * bd + y] = k[j * bc * d + x * d + y];
                vj[x * bd + y] = v[j * bc * d + x * d + y];
            }
        }
        __syncthreads();
        qk_dot_and_scalar<bc, br, bd, num_warps, threads_per_warp>(sij, qi, kj, d, scalar);
#ifndef NO_ROWMAX
        __syncthreads();
        row_max<bc, br, bd, num_warps, threads_per_warp>(mij1, sij, mij0, n);
#endif  // NO_ROWMAX
        __syncthreads();
        minus_max_and_exp<bc, br, bd, num_warps, threads_per_warp>(pij, sij, mij1);
        __syncthreads();
        row_sum<bc, br, bd, num_warps, threads_per_warp>(lij1, pij, lij0, mij0, mij1, n);
        __syncthreads();
        inner_update_o<bc, br, bd, num_warps, threads_per_warp>(oi, pij, vj, mij0, mij1, n, d);
#ifndef NO_ROWMAX
        tmpptr = mij0;
        mij0 = mij1;
        mij1 = tmpptr;
#endif  // NO_ROWMAX
        tmpptr = lij0;
        lij0 = lij1;
        lij1 = tmpptr;
        __syncthreads();
    }
    outer_update_lo<bc, br, bd, num_warps, threads_per_warp>(lij1, oi, mij0, lij0, d);
    __syncthreads();
    // Save O, l, m back to global memory
    for (int y = ty; y < br; y += threads_per_warp) {
        for (int x = tx; x < d; x += num_warps) {
            o[y * d + x] = oi[y * bd + x];
        }
    }
    if (threadIdx.x < br) {
        l[threadIdx.x] = lij1[threadIdx.x];
    }
}

template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void qk_dot_and_scalar(float *out, float *q, float *k, int d, float scalar) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < br; y += threads_per_warp) {
        for (int x = tx; x < bc; x += num_warps) {
            float sum = 0.0F;
            for (int t = 0; t < d; t++) {
                sum += q[y * bd + t] * k[x * bd + t];
            }
            out[y * bc + x] = sum * scalar;
        }
    }
}

template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_max(float *mij1, float *sij, float *mij0, int n) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    if (tx == 0) {
        for (int y = ty; y < br; y += threads_per_warp) {
            float mx = mij0[y];
            for (int t = 0; t < n; t++) {
                mx = fmaxf(mx, sij[y * bc + t]);
            }
            mij1[y] = mx;
        }
    }
}

template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void minus_max_and_exp(float *pij, float *sij, float *mij1) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < br; y += threads_per_warp) {
        for (int x = tx; x < bc; x += num_warps) {
#ifndef NO_ROWMAX
            pij[y * bc + x] = expf(sij[y * bc + x] - mij1[y]);
#else
            pij[y * bc + x] = expf(sij[y * bc + x]);
#endif  // NO_ROWMAX
        }
    }
}

template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_sum(float *lij1, float *pij, float *lij0, float *mij0, float *mij1, int n) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    if (tx == 0) {
        for (int y = ty; y < br; y += threads_per_warp) {
#ifndef NO_ROWMAX
            float sum = expf(mij0[y] - mij1[y]) * lij0[y];
#else
            float sum = lij0[y];
#endif  // NO_ROWMAX
            for (int t = 0; t < n; t++) {
                sum += pij[y * bc + t];
            }
            lij1[y] = sum;
        }
    }
}

template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void inner_update_o(float *oi, float *pij, float *vj, float *mij0, float *mij1, int n, int d) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < br; y += threads_per_warp) {
#ifndef NO_ROWMAX
        float val0 = expf(mij0[y] - mij1[y]);
#else
        float val0 = 1.0F;
#endif  // NO_ROWMAX

        for (int x = tx; x < d; x += num_warps) {
            float sum = 0.0F;
            for (int t = 0; t < n; t++) {
                sum += pij[y * bc + t] * vj[t * bd + x];
            }
            oi[y * bd + x] = val0 * oi[y * bd + x] + sum;
        }
    }
}

template <int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void outer_update_lo(float *lij1, float *oi, float *mij0, float *lij0, int d) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < br; y += threads_per_warp) {
        for (int x = tx; x < d; x += num_warps) {
            oi[y * bd + x] /= lij0[y];
        }
        if (tx == 0) {
#ifndef NO_ROWMAX
            lij1[y] = mij0[y] + logf(lij0[y]);
#else
            lij1[y] = logf(lij0[y]);
#endif  // NO_ROWMAX
        }
    }
}
};  // namespace flash_attention