#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <sys/time.h>
#include <unistd.h>

#include <cassert>
#include <cfloat>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <numeric>

#ifdef PROFILING
#include <nvtx3/nvtx3.hpp>
#define NVTX_RANGE_START(arg) \
    nvtxRangePushA(#arg);
#define NVTX_RANGE_END() \
    nvtxRangePop();
#define NVTX_RANGE_FUNC() \
    NVTX3_FUNC_RANGE()
#else
#define NVTX_RANGE_START(arg) \
    {}
#define NVTX_RANGE_END() \
    {}
#define NVTX_RANGE_FUNC() \
    {}
#endif  // PROFILING

#define CUDA_CHECK(condition)                                                                                     \
    if ((condition) != hipSuccess) {                                                                             \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__, __LINE__); \
        exit(1);                                                                                                  \
    }

struct Data {
    char *input_filename;
    char *output_filename;
    FILE *input_file;
    FILE *output_file;
    int B, N, d;
    float *O;
};

template <typename T>
void cuda_init_array(T *arr, size_t size, T val, hipStream_t stream);
template <typename T>
__global__ void cuda_init_array_kernel(T *arr, size_t size, T val);

namespace flash_attention {
void flash_attention_switch(Data *data);
template <int bc, int br, int cr, int bb>
void flash_attention(Data *data);
template <int bc, int br, int cr>
__global__ void flash_attention_kernel(float *O, float *Q, float *K, float *V, float *L, int N, int d);
template <int bc, int br>
__device__ void qk_dot_and_scalar(float *pij, float *sij, float *q, float *k, int d, float scalar);
template <int bc, int br>
__device__ void row_sum(float *lij1, float *pij, float *lij0, int n);
template <int bc, int br>
__device__ void inner_update_o(float *oi, float *pij, float *vj, int n, int d);
template <int bc, int br>
__device__ void outer_update_lo(float *lij1, float *oi, float *lij0, int d);
};  // namespace flash_attention

int main(int argc, char *argv[]) {
    NVTX_RANGE_FUNC();
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }

    Data data;

    data.input_filename = argv[1];
    data.output_filename = argv[2];

    flash_attention::flash_attention_switch(&data);

    return 0;
}

template <typename T>
void cuda_init_array(T *arr, size_t size, T val, hipStream_t stream) {
    cuda_init_array_kernel<<<(int)ceil((float)size / 1024), 1024, 0, stream>>>(arr, size, val);
}

template <typename T>
__global__ void cuda_init_array_kernel(T *arr, size_t size, T val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        arr[idx] = val;
    }
}

namespace flash_attention {
void flash_attention_switch(Data *data) {
    data->input_file = fopen(data->input_filename, "rb");
    fread(&data->B, sizeof(int), 1, data->input_file);
    fread(&data->N, sizeof(int), 1, data->input_file);
    fread(&data->d, sizeof(int), 1, data->input_file);
    if (data->d <= 64) {
        flash_attention<32, 32, 1, 20>(data);
    }
    data->output_file = fopen(data->output_filename, "wb");
    fwrite(data->O, sizeof(float), data->B * data->N * data->d, data->output_file);

    fclose(data->input_file);
    fclose(data->output_file);

    hipHostFree(data->O);
}

template <int bc, int br, int cr, int bb>
void flash_attention(Data *data) {
    NVTX_RANGE_FUNC();
    int B = data->B;
    int N = data->N;
    int d = data->d;

    // Create a CUDA stream for asynchronous operations
    int num_streams = (int)ceil((float)B / bb);
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    float *Q, *K, *V, *O;
    hipHostMalloc(&Q, B * N * d * sizeof(float));
    hipHostMalloc(&K, B * N * d * sizeof(float));
    hipHostMalloc(&V, B * N * d * sizeof(float));
    hipHostMalloc(&O, B * N * d * sizeof(float));
    data->O = O;
    for (int i = 0; i < B; i++) {
        fread(Q + i * N * d, sizeof(float), N * d, data->input_file);
        fread(K + i * N * d, sizeof(float), N * d, data->input_file);
        fread(V + i * N * d, sizeof(float), N * d, data->input_file);
    }

    float *d_Q, *d_K, *d_V, *d_O;
    float *d_L;
    hipMalloc(&d_Q, B * N * d * sizeof(float));
    hipMalloc(&d_K, B * N * d * sizeof(float));
    hipMalloc(&d_V, B * N * d * sizeof(float));
    hipMalloc(&d_O, B * N * d * sizeof(float));
    hipMalloc(&d_L, B * N * sizeof(float));

    // Kernel launch
    const int smem_size = (br * d +
                           br * d +
                           bc * d +
                           bc * d +
                           br +
                           br +
                           br * bc +
                           br * bc) *
                          sizeof(float);

    NVTX_RANGE_START(flash_attention_execute);
    NVTX_RANGE_START(flash_attention_declare);
    for (int i = 0; i < num_streams; i++) {
        int num_batches = min(bb, B - i * bb);

        // Asynchronous memory copy and initialization
        hipMemcpyAsync(d_Q + i * bb * N * d, Q + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_K + i * bb * N * d, K + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_V + i * bb * N * d, V + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        // Kernel launch
        dim3 grid((int)ceilf((float)N / (br * cr)), num_batches);
        dim3 block(bc, br);
        flash_attention_kernel<bc, br, cr><<<grid, block, smem_size, streams[i]>>>(
            d_O + i * bb * N * d,
            d_Q + i * bb * N * d,
            d_K + i * bb * N * d,
            d_V + i * bb * N * d,
            d_L + i * bb * N,
            N, d);

        // Asynchronous memory copy back to host
        hipMemcpyAsync(O + i * bb * N * d, d_O + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }
    NVTX_RANGE_END();  // flash_attention_declare

    // Synchronize the stream to make sure all operations complete
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
    }
    NVTX_RANGE_END();  // flash_attention_execute

    // Clean up
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_O);
    hipFree(d_L);

    hipHostFree(Q);
    hipHostFree(K);
    hipHostFree(V);
}

template <int bc, int br, int cr>
__global__ void flash_attention_kernel(float *O, float *Q, float *K, float *V, float *L, int N, int d) {
    // Thread and block index
    const int ty = threadIdx.y;
    const int tx = threadIdx.x;
    const int tc = (int)ceilf((float)N / bc);

    // Shared memory allocation
    extern __shared__ float shared_mem[];
    float *oi = shared_mem;      // (br, d)
    float *qi = oi + br * d;     // (br, d)
    float *kj = qi + br * d;     // (bc, d)
    float *vj = kj + bc * d;     // (bc, d)
    float *lij0 = vj + bc * d;   // (br)
    float *lij1 = lij0 + br;     // (br)
    float *sij = lij1 + br;      // (br, bc)
    float *pij = sij + br * bc;  // (br, bc)

    float *tmpptr;

    // Pointer to global memory
    float *o = O + blockIdx.y * N * d + blockIdx.x * cr * br * d;  // (cr, br, d)
    float *q = Q + blockIdx.y * N * d + blockIdx.x * cr * br * d;  // (cr, br, d)
    float *k = K + blockIdx.y * N * d;                             // (N, d)
    float *v = V + blockIdx.y * N * d;                             // (N, d)
    float *l = L + blockIdx.y * N + blockIdx.x * cr * br;          // (cr, br)

    float scalar = 1.0 / sqrtf(d);

    // Load O, Q, l, m to shared memory
    for (int x = tx; x < d; x += bc) {
        oi[ty * d + x] = 0;
        qi[ty * d + x] = q[ty * d + x];
    }
    if (tx == 0) {
        lij0[ty] = 0;
    }
    for (int j = 0; j < tc; j++) {
        int n = min(N - j * bc, bc);
        // Load K and V to shared memory
        for (int y = ty; y < d; y += br) {
            kj[tx * d + y] = k[j * bc * d + tx * d + y];
            vj[tx * d + y] = v[j * bc * d + tx * d + y];
        }
        __syncthreads();
        qk_dot_and_scalar<bc, br>(pij, sij, qi, kj, d, scalar);
        __syncthreads();
        row_sum<bc, br>(lij1, pij, lij0, n);
        __syncthreads();
        inner_update_o<bc, br>(oi, pij, vj, n, d);
        tmpptr = lij0;
        lij0 = lij1;
        lij1 = tmpptr;
        __syncthreads();
    }
    outer_update_lo<bc, br>(lij1, oi, lij0, d);
    // Save O, l, m back to global memory
    for (int x = tx; x < d; x += bc) {
        o[ty * d + x] = oi[ty * d + x];
    }
    if (tx == 0) {
        l[ty] = lij1[ty];
    }
}

template <int bc, int br>
__device__ void qk_dot_and_scalar(float *pij, float *sij, float *q, float *k, int d, float scalar) {
    const int y = threadIdx.y;
    const int x = threadIdx.x;
    float sum = 0.0F;
    for (int t = 0; t < d; t++) {
        sum += q[y * d + t] * k[x * d + t];
    }
    sum *= scalar;
    sij[y * bc + x] = sum;
    pij[y * bc + x] = expf(sum);
}

template <int bc, int br>
__device__ void row_sum(float *lij1, float *pij, float *lij0, int n) {
    if (threadIdx.x == 0) {
        const int y = threadIdx.y;
        float sum = lij0[y];
        for (int t = 0; t < n; t++) {
            sum += pij[y * bc + t];
        }
        lij1[y] = sum;
    }
}

template <int bc, int br>
__device__ void inner_update_o(float *oi, float *pij, float *vj, int n, int d) {
    const int y = threadIdx.y;

    for (int x = threadIdx.x; x < d; x += bc) {
        float sum = 0.0F;
        for (int t = 0; t < n; t++) {
            sum += pij[y * bc + t] * vj[t * d + x];
        }
        oi[y * d + x] = oi[y * d + x] + sum;
    }
}

template <int bc, int br>
__device__ void outer_update_lo(float *lij1, float *oi, float *lij0, int d) {
    const int y = threadIdx.y;

    for (int x = threadIdx.x; x < d; x += bc) {
        oi[y * d + x] /= lij0[y];
    }
    if (threadIdx.x == 0) {
        lij1[y] = logf(lij0[y]);
    }
}
};  // namespace flash_attention