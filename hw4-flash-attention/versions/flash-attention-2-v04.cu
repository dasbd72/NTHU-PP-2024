#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <sys/time.h>
#include <unistd.h>

#include <cassert>
#include <cfloat>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <numeric>

#ifdef PROFILING
#include <nvtx3/nvtx3.hpp>
#define NVTX_RANGE_START(arg) \
    nvtxRangePushA(#arg);
#define NVTX_RANGE_END() \
    nvtxRangePop();
#define NVTX_RANGE_FUNC() \
    NVTX3_FUNC_RANGE()
#else
#define NVTX_RANGE_START(arg) \
    {}
#define NVTX_RANGE_END() \
    {}
#define NVTX_RANGE_FUNC() \
    {}
#endif  // PROFILING

#define CUDA_CHECK(condition)                                                                                     \
    if ((condition) != hipSuccess) {                                                                             \
        fprintf(stderr, "CUDA error: %s at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__, __LINE__); \
        exit(1);                                                                                                  \
    }

constexpr float FLOAT_MIN = -FLT_MAX;

struct Data {
    char *input_filename;
    char *output_filename;
    FILE *input_file;
    FILE *output_file;
    int B, N, d;
    float *O;
};

template <typename T>
void cuda_init_array(T *arr, size_t size, T val, hipStream_t stream);
template <typename T>
__global__ void cuda_init_array_kernel(T *arr, size_t size, T val);

namespace flash_attention {
void flash_attention_switch(Data *data);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
void flash_attention(Data *data);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__global__ void flash_attention_kernel(float *O, float *QKV, float *L, int N, int d);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void qk_dot_and_scalar(float *out, float *q, float *k, int d, float scalar);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_max(float *mij1, float *sij, float *mij0, int n);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void minus_max_and_exp(float *pij, float *sij, float *mij1);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_sum(float *lij1, float *pij, float *lij0, float *mij0, float *mij1, int n);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void inner_update_o(float *oi, float *pij, float *vj, float *mij0, float *mij1, int n, int d);
template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void outer_update_lo(float *lij1, float *oi, float *mij0, float *lij0, int d);
};  // namespace flash_attention

int main(int argc, char *argv[]) {
    NVTX_RANGE_FUNC();
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }

    Data data;

    data.input_filename = argv[1];
    data.output_filename = argv[2];

#ifdef PROFILING
    timespec ts;
    double start, end;
    clock_gettime(CLOCK_REALTIME, &ts);
    start = ts.tv_sec + ts.tv_nsec * 1e-9;
#endif  // PROFILING
    flash_attention::flash_attention_switch(&data);
#ifdef PROFILING
    clock_gettime(CLOCK_REALTIME, &ts);
    end = ts.tv_sec + ts.tv_nsec * 1e-9;
    fprintf(stderr, "took: %lf\n", end - start);
#endif  // PROFILING

    return 0;
}

template <typename T>
void cuda_init_array(T *arr, size_t size, T val, hipStream_t stream) {
    cuda_init_array_kernel<<<(int)ceil((float)size / 1024), 1024, 0, stream>>>(arr, size, val);
}

template <typename T>
__global__ void cuda_init_array_kernel(T *arr, size_t size, T val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        arr[idx] = val;
    }
}

namespace flash_attention {
void flash_attention_switch(Data *data) {
    data->input_file = fopen(data->input_filename, "rb");
    data->output_file = fopen(data->output_filename, "wb");
    fread(&data->B, sizeof(int), 1, data->input_file);
    fread(&data->N, sizeof(int), 1, data->input_file);
    fread(&data->d, sizeof(int), 1, data->input_file);
    if (data->d <= 32) {
        flash_attention<32, 32, 37, 32, 37, 8, 16>(data);
    } else if (data->d <= 64) {
        flash_attention<32, 32, 37, 32, 69, 8, 32>(data);
    }

    fclose(data->input_file);
    fclose(data->output_file);

#ifndef NO_FINALIZE
    hipHostFree(data->O);
#endif  // NO_FINALIZE
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
void flash_attention(Data *data) {
    NVTX_RANGE_FUNC();
    int B = data->B;
    int N = data->N;
    int d = data->d;
#ifdef PROFILING
    fprintf(stderr, "B: %d, N: %d, d: %d\n", B, N, d);
#endif  // PROFILING
    int bb = (int)ceilf((float)B / 64);

    // Create a CUDA stream for asynchronous operations
    int num_streams = (int)ceil((float)B / bb);
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    float *QKV, *O;
    hipHostMalloc(&QKV, B * N * d * 3 * sizeof(float));
    hipHostMalloc(&O, B * N * d * sizeof(float));
    data->O = O;

    float *d_QKV, *d_O;
    float *d_L;
    hipMalloc(&d_QKV, B * N * d * 3 * sizeof(float));
    hipMalloc(&d_O, B * N * d * sizeof(float));
    hipMalloc(&d_L, B * N * sizeof(float));

    // Kernel launch
    const int smem_size = (br * bd +
                           br * bd +
                           bc * bd +
                           bc * bd +
                           br +
                           br +
                           br +
                           br +
                           br * bc +
                           br * bc) *
                          sizeof(float);

    NVTX_RANGE_START(flash_attention_execute);
    NVTX_RANGE_START(flash_attention_declare);
    for (int i = 0; i < num_streams; i++) {
        int num_batches = min(bb, B - i * bb);

        // Load data to host memory
        fread(QKV + i * bb * N * d * 3, sizeof(float), num_batches * N * d * 3, data->input_file);

        // Asynchronous memory copy and initialization
        hipMemcpyAsync(d_QKV + i * bb * N * d * 3, QKV + i * bb * N * d * 3, num_batches * N * d * 3 * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        // Kernel launch
        dim3 grid((int)ceilf((float)N / ar), num_batches);
        dim3 block(num_warps * threads_per_warp);
        flash_attention_kernel<ac, ar, bc, br, bd, num_warps, threads_per_warp><<<grid, block, smem_size, streams[i]>>>(
            d_O + i * bb * N * d,
            d_QKV + i * bb * N * d * 3,
            d_L + i * bb * N,
            N, d);

        // Asynchronous memory copy back to host
        hipMemcpyAsync(O + i * bb * N * d, d_O + i * bb * N * d, num_batches * N * d * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }
    NVTX_RANGE_END();  // flash_attention_declare

    // Synchronize the stream to make sure all operations complete
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(streams[i]);
        fwrite(data->O + i * bb * N * d, sizeof(float), bb * N * d, data->output_file);
    }
    NVTX_RANGE_END();  // flash_attention_execute

    // Clean up
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }

#ifndef NO_FINALIZE
    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_O);
    hipFree(d_L);

    hipHostFree(Q);
    hipHostFree(K);
    hipHostFree(V);
#endif  // NO_FINALIZE
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__global__ void flash_attention_kernel(float *O, float *QKV, float *L, int N, int d) {
    // Thread and block index
    const int tx = threadIdx.x % num_warps;
    const int ty = threadIdx.x / num_warps;
    const int tc = (int)ceilf((float)N / ac);

    // Shared memory allocation
    extern __shared__ float shared_mem[];
    float *oi = shared_mem;      // (br, bd)
    float *qi = oi + br * bd;    // (br, bd)
    float *kj = qi + br * bd;    // (bc, bd)
    float *vj = kj + bc * bd;    // (bc, bd)
    float *lij0 = vj + bc * bd;  // (br)
    float *lij1 = lij0 + br;     // (br)
    float *mij0 = lij1 + br;     // (br)
    float *mij1 = mij0 + br;     // (br)
    float *sij = mij1 + br;      // (br, bc)
    float *pij = sij + br * bc;  // (br, bc)

    float *tmpptr;

    // Pointer to global memory
    float *o = O + blockIdx.y * N * d + blockIdx.x * ar * d;        // (ar, d)
    float *q = QKV + blockIdx.y * N * d * 3 + blockIdx.x * ar * d;  // (ar, d)
    float *k = QKV + blockIdx.y * N * d * 3 + N * d * 1;            // (N, d)
    float *v = QKV + blockIdx.y * N * d * 3 + N * d * 2;            // (N, d)
    float *l = L + blockIdx.y * N + blockIdx.x * ar;                // (ar)

    float scalar = 1.0 / sqrtf(d);

    // Load O, Q, l, m to shared memory
    for (int y = ty; y < ar; y += threads_per_warp) {
        for (int x = tx; x < d; x += num_warps) {
            oi[y * bd + x] = 0;
            qi[y * bd + x] = q[y * d + x];
        }
    }
    if (threadIdx.x < ar) {
        lij0[threadIdx.x] = 0;
#ifndef NO_ROWMAX
        mij0[threadIdx.x] = FLOAT_MIN;
#endif  // NO_ROWMAX
    }
    for (int j = 0; j < tc; j++) {
        int n = min(N - j * ac, ac);
        // Load K and V to shared memory
        for (int x = tx; x < n; x += num_warps) {
            for (int y = ty; y < d; y += threads_per_warp) {
                kj[x * bd + y] = k[j * ac * d + x * d + y];
                vj[x * bd + y] = v[j * ac * d + x * d + y];
            }
        }
        __syncthreads();
        qk_dot_and_scalar<ac, ar, bc, br, bd, num_warps, threads_per_warp>(sij, qi, kj, d, scalar);
#ifndef NO_ROWMAX
        __syncthreads();
        row_max<ac, ar, bc, br, bd, num_warps, threads_per_warp>(mij1, sij, mij0, n);
#endif  // NO_ROWMAX
        __syncthreads();
        minus_max_and_exp<ac, ar, bc, br, bd, num_warps, threads_per_warp>(pij, sij, mij1);
        __syncthreads();
        row_sum<ac, ar, bc, br, bd, num_warps, threads_per_warp>(lij1, pij, lij0, mij0, mij1, n);
        inner_update_o<ac, ar, bc, br, bd, num_warps, threads_per_warp>(oi, pij, vj, mij0, mij1, n, d);
#ifndef NO_ROWMAX
        tmpptr = mij0;
        mij0 = mij1;
        mij1 = tmpptr;
#endif  // NO_ROWMAX
        tmpptr = lij0;
        lij0 = lij1;
        lij1 = tmpptr;
        __syncthreads();
    }
    outer_update_lo<ac, ar, bc, br, bd, num_warps, threads_per_warp>(lij1, oi, mij0, lij0, d);
    __syncthreads();
    // Save O, l, m back to global memory
    for (int y = ty; y < ar; y += threads_per_warp) {
        for (int x = tx; x < d; x += num_warps) {
            o[y * d + x] = oi[y * bd + x];
        }
    }
    if (threadIdx.x < ar) {
        l[threadIdx.x] = lij1[threadIdx.x];
    }
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void qk_dot_and_scalar(float *out, float *q, float *k, int d, float scalar) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < ar; y += threads_per_warp) {
        for (int x = tx; x < ac; x += num_warps) {
            float sum = 0.0F;
            for (int t = 0; t < d; t++) {
                sum += q[y * bd + t] * k[x * bd + t];
            }
            out[y * bc + x] = sum * scalar;
        }
    }
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_max(float *mij1, float *sij, float *mij0, int n) {
    for (int y = threadIdx.x; y < ar; y += blockDim.x) {
        float mx = mij0[y];
        for (int t = 0; t < n; t++) {
            mx = fmaxf(mx, sij[y * bc + t]);
        }
        mij1[y] = mx;
    }
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void minus_max_and_exp(float *pij, float *sij, float *mij1) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < ar; y += threads_per_warp) {
        float mx = mij1[y];
        for (int x = tx; x < ac; x += num_warps) {
#ifndef NO_ROWMAX
            pij[y * bc + x] = expf(sij[y * bc + x] - mx);
#else
            pij[y * bc + x] = expf(sij[y * bc + x]);
#endif  // NO_ROWMAX
        }
    }
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void row_sum(float *lij1, float *pij, float *lij0, float *mij0, float *mij1, int n) {
    for (int y = threadIdx.x; y < ar; y += blockDim.x) {
#ifndef NO_ROWMAX
        float sum = expf(mij0[y] - mij1[y]) * lij0[y];
#else
        float sum = lij0[y];
#endif  // NO_ROWMAX
        for (int t = 0; t < n; t++) {
            sum += pij[y * bc + t];
        }
        lij1[y] = sum;
    }
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void inner_update_o(float *oi, float *pij, float *vj, float *mij0, float *mij1, int n, int d) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < ar; y += threads_per_warp) {
#ifndef NO_ROWMAX
        float val0 = expf(mij0[y] - mij1[y]);
#else
        float val0 = 1.0F;
#endif  // NO_ROWMAX

        for (int x = tx; x < d; x += num_warps) {
            float sum = 0.0F;
            for (int t = 0; t < n; t++) {
                sum += pij[y * bc + t] * vj[t * bd + x];
            }
            oi[y * bd + x] = val0 * oi[y * bd + x] + sum;
        }
    }
}

template <int ac, int ar, int bc, int br, int bd, int num_warps, int threads_per_warp>
__device__ __forceinline__ void outer_update_lo(float *lij1, float *oi, float *mij0, float *lij0, int d) {
    int tx = threadIdx.x % num_warps;
    int ty = threadIdx.x / num_warps;
    for (int y = ty; y < ar; y += threads_per_warp) {
        for (int x = tx; x < d; x += num_warps) {
            oi[y * bd + x] /= lij0[y];
        }
        if (tx == 0) {
#ifndef NO_ROWMAX
            lij1[y] = mij0[y] + logf(lij0[y]);
#else
            lij1[y] = logf(lij0[y]);
#endif  // NO_ROWMAX
        }
    }
}
};  // namespace flash_attention